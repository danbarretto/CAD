#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define THREADS 10

// BOTA GLOBAL
__global__ void prodEscalar(int* A, int* B, int* prodEsc, int* somaDosProd, int dim) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if(i < dim && j < dim ){
    int produto = A[i*dim + j] * B[i*dim + j];
    atomicAdd(prodEsc+i, produto); 
    atomicAdd(somaDosProd, produto);
  }
}

__global__ void soma_elementos(int *vetorA,int *soma,int tam){
    //Calcula o índice global da thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < tam){
        //Faz a soma entre elemento do vetor no índice idx e o conteúdo de soma
        atomicAdd(soma,vetorA[idx]); 
    }  
}

__global__ void min_max_elementos(int *A, int *B, int *max_comp, int tam)
{
    //Calcula o índice global da thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < tam)
    {
        //Faz a soma entre elemento do vetor no índice idx e o conteúdo de soma
        int local_max = A[i*dim + j] > B[i*dim + j] ? A : B;
        int local_min = A[i*dim + j] < B[i*dim + j] ? A : B;
        
        atomicMax(max_comp, local_max);
        atomicMin(max_comp, local_min);
    }
}

// Matriz A e B na memoria global
// 

int main(int argc, char **argv) {
  // Declara as matrizes
  int *A, *B;

  // Declara as variáveis de índice
  int i, j, k, dim;
  // Declara o acumulador para o produto escalar global
  int somape, minimo, maximo;

  // Declara um vetor para os produtos escalares locais
  int *prod_escalar;

  FILE *inputfile;      // handler para o arquivo de entrada
  char *inputfilename;  // nome do arquivo de entrada

  if (argc < 2) {
    printf("Please run with input file name, i.e., num_perf_mpi inputfile.ext\n");
    exit(-1);
  }

  inputfilename = (char *)malloc(256 * sizeof(char));
  strcpy(inputfilename, argv[1]);

  printf("inputfilename=%s\n", inputfilename);
  fflush(0);

  if ((inputfile = fopen(inputfilename, "r")) == 0) {
    printf("Error openning input file.\n");
    exit(-1);
  }

  fscanf(inputfile, "%d\n", &dim);  // Lê a dimensão das matrizes

  
  // Aloca as matrizes
  A = (int *)malloc(dim * dim * sizeof(int));
  B = (int *)malloc(dim * dim * sizeof(int));

  // Aloca um vetor para armazenar os produtos escalares de cada linha
  prod_escalar = (int *)malloc(dim * sizeof(int));

  // Lê a matriz A
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      fscanf(inputfile, "%d ", &(A[i * dim + j]));
    }
  }

  // Lê a matriz B
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      fscanf(inputfile, "%d ", &(B[i * dim + j]));
    }
  }

  // fecha o arquivo de entrada
  fclose(inputfile);

  int *A_d, *B_d;
  //aloca matrizes na gpu
  hipMalloc(&A_d,dim * dim * sizeof(int));
  hipMalloc(&B_d,dim * dim * sizeof(int));

  hipMemcpy(A, A_d, dim*dim*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(B, B_d, dim*dim*sizeof(int), hipMemcpyDeviceToHost);

  int *min_D, *max_D, *prod_D, *soma_D;
  hipMalloc(&min_D, sizeof(int));
  hipMalloc(&max_D, sizeof(int));  
  hipMalloc(&prod_D, sizeof(int));  
  hipMalloc(&soma_D, sizeof(int));  
  
  //Define a quantidade de threads por bloco
  dim3 threadsPerBlock(THREADS,THREADS);
  //Define a quantidade de blocos por grade
  dim3 blocksPerGrid((dim+(threadsPerBlock.x-1)) / threadsPerBlock.x, (dim+(threadsPerBlock.y-1)) / threadsPerBlock.y);

  min_max_elementos<<blocksPerGridb, threadsPerBlock>>(A_d, B_d, min_D, dim);
  
  prodEscalar<<blocksPerGridb, threadsPerBlock>>(A_d, B_d, prod_D, min_D, soma_D, dim);



  /*
  // Calcula o produto escalar de cada linha
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      prod_escalar[j] += A[i * dim + j] * B[i * dim + j];
    }
  }

  // Acumula os produtos das linhas (faz o produto escalar global)
  somape = 0;
  for (i = 0; i < dim; i++) {
    somape += prod_escalar[i];
  }

  // encontra o mínimo da matriz A
  minimo = A[0];
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      if (A[i * dim + j] < minimo) {
        minimo = A[i * dim + j];
      }
    }
  }
  // encontra o mínimo da matriz B
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      if (B[i * dim + j] < minimo) {
        minimo = B[i * dim + j];
      }
    }
  }

  // encontra o máximo da matriz A
  maximo = A[0];
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      if (A[i * dim + j] > maximo) {
        maximo = A[i * dim + j];
      }
    }
  }
  // encontra o máximo da matriz B
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      if (B[i * dim + j] > maximo) {
        maximo = B[i * dim + j];
      }
    }
  }
  */

 
 
  // Imprime o resultado
  printf("%d %d %d\n", somape, minimo, maximo);

  // Libera as matrizes
  free(A);
  free(B);
  hipFree(B_d);
  hipFree(A_d);
  hipFree(prod_escalar_d);
  // Libera o vetor
  free(prod_escalar);
}
