/*
  Daniel Sá Barretto Prado Garcia 10374344
  Tiago Marino Silva 10734748
  Felipe Guilermmo Santuche Moleiro 10724010
  Laura Genari Alves de Jesus 10801180
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define THREADS 32
#define INF 0x7fffffff

__global__ void prodEscalar(int* A, int* B, int* somaDosProd, int dim) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if(i < dim && j < dim ){
    int produto = A[i*dim + j] * B[i*dim + j];
    atomicAdd(somaDosProd, produto);
  }
}

__global__ void min_max_elementos(int* A, int* B, int* max_comp, int* min_comp, int dim)
{
    //Calcula o índice global da thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < dim && j < dim)
    {
        //Faz a soma entre elemento do vetor no índice idx e o conteúdo de soma
        int local_max = A[i*dim + j] > B[i*dim + j] ? A[i*dim + j] : B[i*dim + j];
        int local_min = A[i*dim + j] < B[i*dim + j] ? A[i*dim + j] : B[i*dim + j];
        
        atomicMax(max_comp, local_max);
        atomicMin(min_comp, local_min);
    }
}

int main(int argc, char **argv) 
{
  // Declara as matrizes
  int *A, *B;

  // Declara as variáveis de índice
  int i, j, dim;

  FILE *inputfile;      // handler para o arquivo de entrada
  char *inputfilename;  // nome do arquivo de entrada

  if (argc < 2) {
    printf("Please run with input file name, i.e., num_perf_mpi inputfile.ext\n");
    exit(-1);
  }

  inputfilename = (char *)malloc(256 * sizeof(char));
  strcpy(inputfilename, argv[1]);

  if ((inputfile = fopen(inputfilename, "r")) == 0) {
    printf("Error openning input file.\n");
    exit(-1);
  }

  fscanf(inputfile, "%d\n", &dim);  // Lê a dimensão das matrizes

  
  // Aloca as matrizes
  A = (int *)malloc(dim * dim * sizeof(int));
  B = (int *)malloc(dim * dim * sizeof(int));


  // Lê a matriz A
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      fscanf(inputfile, "%d ", &(A[i * dim + j]));
    }
  }

  // Lê a matriz B
  for (i = 0; i < dim; i++) {
    for (j = 0; j < dim; j++) {
      fscanf(inputfile, "%d ", &(B[i * dim + j]));
    }
  }

  // fecha o arquivo de entrada
  fclose(inputfile);

  int *A_d, *B_d;
  //aloca matrizes na gpu
  hipMalloc(&A_d,dim * dim * sizeof(int));
  hipMalloc(&B_d,dim * dim * sizeof(int));

  hipMemcpy(A_d, A, dim*dim*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, dim*dim*sizeof(int), hipMemcpyHostToDevice);
  
  int max = -INF, min = INF, soma = 0;
  int *min_D, *max_D, *soma_D;
  hipMalloc(&min_D, sizeof(int));
  hipMalloc(&max_D, sizeof(int));  
  hipMalloc(&soma_D, sizeof(int));
  
  hipMemcpy(max_D, &max, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(min_D, &min, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(soma_D, &soma, sizeof(int), hipMemcpyHostToDevice);
  
  //Define a quantidade de threads por bloco
  dim3 threadsPerBlock(THREADS,THREADS);
  //Define a quantidade de blocos por grade
  dim3 blocksPerGrid((dim+(threadsPerBlock.x-1)) / threadsPerBlock.x, (dim+(threadsPerBlock.y-1)) / threadsPerBlock.y);

  min_max_elementos<<<blocksPerGrid, threadsPerBlock>>>(A_d, B_d, max_D, min_D, dim);
  
  prodEscalar<<<blocksPerGrid, threadsPerBlock>>>(A_d, B_d, soma_D, dim);

  
  //Copia o resultado da soma de volta para o host
  //cudaMemcpy(prod_escalar, prod_D, dim * sizeof(int), cudaMemcpyDeviceToHost);
  hipMemcpy(&max, max_D,sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&min, min_D,sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&soma, soma_D,sizeof(int), hipMemcpyDeviceToHost);
 
  //Imprime o resultado
  printf("%d %d %d\n", soma, min, max); 

  // Liberação de memória alocada
  free(A);
  free(B);
  hipFree(B_d);
  hipFree(A_d);
  hipFree(min_D);
  hipFree(max_D);
  hipFree(soma_D);
 
}